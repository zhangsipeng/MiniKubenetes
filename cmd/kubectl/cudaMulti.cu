#include "hip/hip_runtime.h"
#include <stdio.h>
struct Matrix
{
    int width;
    int height;
    int *elements;
};

__device__ int getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
};


__device__ void setElement(Matrix *A, int row, int col, int value)
{
	A->elements[row * A->width + col] = value;
};

__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		Cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, Cvalue);
}


int main()
{
    int width = 32;
    int height = 32;
    Matrix *A, *B, *C;
    // 
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(int);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1;
        B->elements[i] = 2;
    }

    // 
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    // 
    matMulKernel << < gridSize, blockSize >> >(A, B, C);


    // 
    hipDeviceSynchronize();
    // 
    for (int i=0;i<height;i++){
        for (int j=0;j<width;j++){
            printf("%d\t",C->elements[i*width+j]);
        }
        printf("\n");
    }

    return 0;
}