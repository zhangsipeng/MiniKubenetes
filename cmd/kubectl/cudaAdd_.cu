#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
 
#include <stdio.h>
#include <math.h>
#define Row  8
#define Col 4
 
 
__global__ void addKernel(int **C,  int **A)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;
	if (idx < Col && idy < Row) {
		C[idy][idx] = A[idy][idx] + 10;
	}
}
 
int main()
{
	int **A = (int **)malloc(sizeof(int*) * Row);
	int **C = (int **)malloc(sizeof(int*) * Row);
	int *dataA = (int *)malloc(sizeof(int) * Row * Col);
	int *dataC = (int *)malloc(sizeof(int) * Row * Col);
	int **d_A;
	int **d_C;
	int *d_dataA;
	int *d_dataC;
    //malloc device memory
	hipMalloc((void**)&d_A, sizeof(int **) * Row);
	hipMalloc((void**)&d_C, sizeof(int **) * Row);
	hipMalloc((void**)&d_dataA, sizeof(int) *Row*Col);
	hipMalloc((void**)&d_dataC, sizeof(int) *Row*Col);
	//set value
	for (int i = 0; i < Row*Col; i++) {
		dataA[i] = i+1;
	}

	for (int i = 0; i < Row; i++) {
		A[i] = d_dataA + Col * i;
		C[i] = d_dataC + Col * i;
	}
	
	hipMemcpy(d_A, A, sizeof(int*) * Row, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(int*) * Row, hipMemcpyHostToDevice);
	hipMemcpy(d_dataA, dataA, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
	dim3 threadPerBlock(4, 4);
	dim3 blockNumber( (Col + threadPerBlock.x - 1)/ threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y );
	printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
	addKernel << <blockNumber, threadPerBlock >> > (d_C, d_A);

	hipMemcpy(dataC, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);
 
	for (int i = 0; i < Row*Col; i++) {
		if (i%Col == 0) {
			printf("\n");
		}
		printf("%d\t", dataC[i]);
	}
	printf("\n");
    
}
